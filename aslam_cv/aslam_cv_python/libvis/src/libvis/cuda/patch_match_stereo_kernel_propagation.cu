#include "hip/hip_runtime.h"
// Copyright 2017, 2019 ETH Zürich, Thomas Schöps
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include "libvis/cuda/patch_match_stereo.cuh"

#include <hip/hip_math_constants.h>

#include "libvis/cuda/cuda_auto_tuner.h"
#include "libvis/cuda/cuda_unprojection_lookup.cuh"
#include "libvis/cuda/cuda_util.cuh"
#include "libvis/cuda/cuda_util.h"
#include "libvis/cuda/patch_match_stereo_cost.cuh"
#include "libvis/cuda/patch_match_stereo_util.cuh"

namespace vis {

__global__ void PatchMatchPropagationStepCUDAKernel(
    StereoParametersSingleCUDA p,
    int match_metric,
    float second_best_min_distance_factor,
    CUDABuffer_<float> best_inv_depth_map) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x >= p.context_radius &&
      y >= p.context_radius &&
      x < p.inv_depth_map.width() - p.context_radius &&
      y < p.inv_depth_map.height() - p.context_radius) {
    // "Pulling" the values inwards.
    float2 nxy = p.reference_unprojection_lookup.UnprojectPoint(x, y);
    
    #pragma unroll
    for (int dy = -1; dy <= 1; ++ dy) {
      #pragma unroll
      for (int dx = -1; dx <= 1; ++ dx) {
        if ((dx == 0 && dy == 0) ||
            (dx != 0 && dy != 0)) {
          continue;
        }
        
        // Compute inv_depth for propagating the pixel at (x + dx, y + dy) to the center pixel.
        float2 other_nxy = p.reference_unprojection_lookup.UnprojectPoint(x + dx, y + dy);
        
        float other_inv_depth = p.inv_depth_map(y + dy, x + dx);
        float other_depth = 1.f / other_inv_depth;
        
        char2 other_normal_xy_char = p.normals(y + dy, x + dx);
        const float2 other_normal_xy = make_float2(
            other_normal_xy_char.x * (1 / 127.f), other_normal_xy_char.y * (1 / 127.f));
        float other_normal_z = -sqrtf(1.f - other_normal_xy.x * other_normal_xy.x - other_normal_xy.y * other_normal_xy.y);
        
        float plane_d = (other_nxy.x * other_depth) * other_normal_xy.x + (other_nxy.y * other_depth) * other_normal_xy.y + other_depth * other_normal_z;
        
        float inv_depth = CalculatePlaneInvDepth2(plane_d, other_normal_xy, other_normal_z, nxy.x, nxy.y);
        
        // Test whether to propagate
        float proposal_costs = ComputeCosts(
            x, y,
            other_normal_xy,
            inv_depth,
            p,
            match_metric,
            second_best_min_distance_factor,
            best_inv_depth_map);
        
        if (!::isnan(proposal_costs) && !(proposal_costs >= p.costs(y, x))) {
          p.costs(y, x) = proposal_costs;
          
          // NOTE: Other threads could read these values while they are written,
          //       but it should not be very severe if that happens.
          //       Could use ping-pong buffers to avoid that.
          p.normals(y, x) = make_char2(other_normal_xy.x * 127.f, other_normal_xy.y * 127.f);
          p.inv_depth_map(y, x) = inv_depth;
        }
      }  // loop over dx
    }  // loop over dy
  }
}

void PatchMatchPropagationStepCUDA(
    const StereoParametersSingle& p,
    int match_metric,
    float second_best_min_distance_factor,
    CUDABuffer_<float>* best_inv_depth_map) {
  CHECK_CUDA_NO_ERROR();
  CUDA_AUTO_TUNE_2D(
      PatchMatchPropagationStepCUDAKernel,
      16, 16,
      p.inv_depth_map.width(), p.inv_depth_map.height(),
      0, p.stream,
      /* kernel parameters */
      StereoParametersSingleCUDA(p),
      match_metric,
      second_best_min_distance_factor,
      best_inv_depth_map ? *best_inv_depth_map : CUDABuffer_<float>());
  CHECK_CUDA_NO_ERROR();
}


__global__ void PatchMatchPropagationStepCUDAKernel(
    StereoParametersMultiCUDA p,
    int match_metric,
    float second_best_min_distance_factor,
    CUDABuffer_<float> best_inv_depth_map) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x >= p.context_radius &&
      y >= p.context_radius &&
      x < p.inv_depth_map.width() - p.context_radius &&
      y < p.inv_depth_map.height() - p.context_radius) {
    // "Pulling" the values inwards.
    float2 nxy = p.reference_unprojection_lookup.UnprojectPoint(x, y);
    
    #pragma unroll
    for (int dy = -1; dy <= 1; ++ dy) {
      #pragma unroll
      for (int dx = -1; dx <= 1; ++ dx) {
        if ((dx == 0 && dy == 0) ||
            (dx != 0 && dy != 0)) {
          continue;
        }
        
        // Compute inv_depth for propagating the pixel at (x + dx, y + dy) to the center pixel.
        float2 other_nxy = p.reference_unprojection_lookup.UnprojectPoint(x + dx, y + dy);
        
        float other_inv_depth = p.inv_depth_map(y + dy, x + dx);
        float other_depth = 1.f / other_inv_depth;
        
        char2 other_normal_xy_char = p.normals(y + dy, x + dx);
        const float2 other_normal_xy = make_float2(
            other_normal_xy_char.x * (1 / 127.f), other_normal_xy_char.y * (1 / 127.f));
        float other_normal_z = -sqrtf(1.f - other_normal_xy.x * other_normal_xy.x - other_normal_xy.y * other_normal_xy.y);
        
        float plane_d = (other_nxy.x * other_depth) * other_normal_xy.x + (other_nxy.y * other_depth) * other_normal_xy.y + other_depth * other_normal_z;
        
        float inv_depth = CalculatePlaneInvDepth2(plane_d, other_normal_xy, other_normal_z, nxy.x, nxy.y);
        
        // Test whether to propagate
        const char2 normal_char = p.normals(y, x);
        if (IsCostOfProposedChangeLower(
            x, y,
            make_float2(normal_char.x * (1 / 127.f), normal_char.y * (1 / 127.f)),
            p.inv_depth_map(y, x),
            other_normal_xy,
            inv_depth,
            p,
            match_metric,
            second_best_min_distance_factor,
            best_inv_depth_map)) {
          // NOTE: Other threads could read these values while they are written,
          //       but it should not be very severe if that happens.
          //       Could use ping-pong buffers to avoid that.
          p.normals(y, x) = make_char2(other_normal_xy.x * 127.f, other_normal_xy.y * 127.f);
          p.inv_depth_map(y, x) = inv_depth;
        }
      }  // loop over dx
    }  // loop over dy
  }  // check for the thread's validity
}

void PatchMatchPropagationStepCUDA(
    const StereoParametersMulti& p,
    int match_metric,
    float second_best_min_distance_factor,
    CUDABuffer_<float>* best_inv_depth_map) {
  CHECK_CUDA_NO_ERROR();
  CUDA_AUTO_TUNE_2D(
      PatchMatchPropagationStepCUDAKernel,
      16, 16,
      p.inv_depth_map.width(), p.inv_depth_map.height(),
      0, p.stream,
      /* kernel parameters */
      StereoParametersMultiCUDA(p),
      match_metric,
      second_best_min_distance_factor,
      best_inv_depth_map ? *best_inv_depth_map : CUDABuffer_<float>());
  CHECK_CUDA_NO_ERROR();
}

}
